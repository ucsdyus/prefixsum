#include "hip/hip_runtime.h"
#include <iostream>
#include <memory>
#include <cstdio>
#include <cstdlib>
#include <cmath>
#include <hip/hip_runtime.h>
#include "helper.h"

// L2
void verify(int N, ValueType* calc, ValueType* gt, ValueType eps);

// pfxsum kernels
void pfxsum_host(int N, ValueType* vals, ValueType* pfx);

void pfxsum_v1_allshare(int N, ValueType* vals, ValueType* pfx);

int main(int argc, char** argv) {
    CmdOptions cmd_opt;
    cmdline(argc, argv, cmd_opt);
    int capability = ReportDevice();
    printf("GPU capacity: %d\n", capability);

    std::unique_ptr<ValueType[]> host_vals(new ValueType[cmd_opt.n]);
    genData(cmd_opt.n, host_vals.get(), cmd_opt.use_rand);

    std::unique_ptr<ValueType[]> ground_truth(new ValueType[cmd_opt.n]);
    pfxsum_host(cmd_opt.n, host_vals.get(), ground_truth.get());

    std::unique_ptr<ValueType[]> res(new ValueType[cmd_opt.n]);

    ValueType* cuda_vals = nullptr;
    ValueType* cuda_pfx = nullptr;
    
    hipMalloc(&cuda_vals, cmd_opt.n * sizeof(ValueType));
    checkCUDAError("Error allocating device memory for values");
    hipMalloc(&cuda_pfx, cmd_opt.n * sizeof(ValueType));
    checkCUDAError("Error allocating device memory for prefix sum");

    hipMemcpy(cuda_vals, host_vals.get(), cmd_opt.n * sizeof(ValueType), hipMemcpyHostToDevice);
    checkCUDAError("Error copying values from host to device");
    hipMemset(cuda_pfx, 0, cmd_opt.n * sizeof(ValueType));
    checkCUDAError("Error set prefix sum to 0");
    
    hipDeviceSynchronize();
    double t_device = -getTime();

    switch(cmd_opt.version) {
        case 0:
            for (int i = 0; i < cmd_opt.reps; ++i) {
                pfxsum_host(cmd_opt.n, host_vals.get(), res.get());
            }
            break;
        case 1:
            for (int i = 0; i < cmd_opt.reps; ++i) {
                pfxsum_v1_allshare(cmd_opt.n, cuda_vals, cuda_pfx);
            }
            break;
        default:
            std::cout << "Not Implemented Error: version " << cmd_opt.version << std::endl;
            exit(-1);
    }

    hipDeviceSynchronize();
    t_device += getTime();

    checkCUDAError("Error in cuda kernel");
    switch(cmd_opt.version) {
        case 1:
        hipMemcpy(res.get(), cuda_pfx, cmd_opt.n * sizeof(ValueType), hipMemcpyDeviceToHost);
        checkCUDAError("Error copying values from device to host");
        hipFree(cuda_vals);
        checkCUDAError("Error releasing values");
        hipFree(cuda_pfx);
        checkCUDAError("Error releasing prefix sum");
        break;
    }

    double gflops_d = getGflops(cmd_opt.n, cmd_opt.reps, t_device);
    printf("Computation time: %f sec. [%f gflops]\n", t_device, gflops_d);
    perfString(t_device, gflops_d, cmd_opt);

    verify(cmd_opt.n, res.get(), ground_truth.get(), cmd_opt.eps);
    return 0;
}

void verify(int N, ValueType* res, ValueType* gt, ValueType eps) {
    ValueType err = 0;
    for (int i = 0; i < N; ++i) {
        err += (res[i] - gt[i]) * (res[i] - gt[i]);
    }
    err = std::sqrt(err);
    if (err > eps) {
        std::cout << "*** a total of error: " << err
            << " exceeds eps: " << eps << std::endl;
        std::cout << res[0];
        for (int i = 1; i < N; ++i) {
            std::cout << " " << res[i];
        }
        std::cout << std::endl;
    } else {
        std::cout << "*** error < eps: " << eps << std::endl;
        std::cout << "*** answer verified" << std::endl;
    }
}